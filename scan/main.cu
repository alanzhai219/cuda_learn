#include <stdio.h> 
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 64

int main(int argc, char* argv[]) {
  const int num = N;
  // init and allocate data
  float * arr_in = (float*)malloc(num * sizeof(float));
  float * arr_out = (float*)malloc(num * sizeof(float));
  for (int i=0; i<num; ++i) {
    arr_in[i] = 1.0;
  }
  for (int i=0; i<num; ++i) {
    arr_out[i] = 0.0;
  }

  // malloc in cuda
  float* dev_in = NULL;
  float* dev_out = NULL;
  hipMalloc((float**)&dev_in, num * sizeof(float));
  hipMalloc((float**)&dev_out, num * sizeof(float));

  dim3 block(32);
  dim3 grid(2);
  // copy from host to device
  hipMemcpy(dev_in, arr_in, num * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_out, arr_out, num * sizeof(float), hipMemcpyHostToDevice);

  // call scan func
  prescan<<<grid, block, sizeof(float)*N*2>>>(dev_out, dev_in, num);

  // sync
  hipDeviceSynchronize();

  hipMemcpy(arr_out, dev_out, num * sizeof(float), hipMemcpyDeviceToHost);

  for (int i=0; i<num; i++) {
    printf("%f\n", arr_out[i]);
  }
  free(arr_in);
  free(arr_out);
  hipFree(dev_in);
  hipFree(dev_out);
  return 0;
}
