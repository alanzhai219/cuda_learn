
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

int main(void)
{
    const int n = 32;
    const size_t sz = size_t(n) * sizeof(int);
    int *dJunk;
    hipMalloc((void**)&dJunk, sz);
    hipMemset(dJunk, 0, sz);
    hipMemset(dJunk, 0x12, 32);

    int *Junk = new int[n];

    hipMemcpy(Junk, dJunk, sz, hipMemcpyDeviceToHost);

    for(int i=0; i<n; i++) {
        // fprintf(stdout, "%d %d\n", i, Junk[i]);
        std::cout << std::hex << Junk[i] << std::endl;
    }

    hipDeviceReset();
    return 0;
}
