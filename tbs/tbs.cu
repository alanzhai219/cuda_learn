
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

#define NB 1000
// increase array length here if your GPU has more than 32 SMs
#define MAX_SM 32
// set HANG_TEST to 1 to demonstrate a hang for test purposes
#define HANG_TEST 0

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

static __device__ __inline__ uint32_t __smid(){
    uint32_t smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    return smid;}

__device__ volatile int blocks_completed = 0;
// increase array length here if your GPU has more than 32 SMs
__device__ int first_SM[MAX_SM];

// launch with one thread per block only
__global__ void tkernel(int num_blocks, int num_SMs){

  int my_SM = __smid();
  int im_not_first = atomicCAS(first_SM+my_SM, 0, 1);
  if (!im_not_first){
    while (blocks_completed < (num_blocks-num_SMs+HANG_TEST));
  }
  atomicAdd((int *)&blocks_completed, 1);
}

int main(int argc, char *argv[]){
  unsigned my_dev = 0;
  if (argc > 1) my_dev = atoi(argv[1]);
  hipSetDevice(my_dev);
  cudaCheckErrors("invalid CUDA device");

  int tot_SM = 0;
  hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, my_dev);
  cudaCheckErrors("CUDA error");
  if (tot_SM > MAX_SM) {
    printf("program configuration error\n");
    return 1;
  }
  printf("running on device %d, with %d SMs\n", my_dev, tot_SM);

  int temp[MAX_SM];
  for (int i = 0; i < MAX_SM; i++)
    temp[i] = 0;

  hipMemcpyToSymbol(HIP_SYMBOL(first_SM), temp, MAX_SM*sizeof(int));
  cudaCheckErrors("hipMemcpyToSymbol fail");
  tkernel<<<NB, 1>>>(NB, tot_SM);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel error");
}
