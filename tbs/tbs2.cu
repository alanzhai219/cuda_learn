#include <stdio.h>

#include <stdlib.h>

#include <hip/hip_runtime.h>

/* E.D. Riedijk */

__device__ uint get_smid(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}

__global__ void kern(int *sm){

   if (threadIdx.x==0)

      sm[blockIdx.x]=get_smid();

}

int main(int argc, char *argv[]){

   int N = atoi(argv[1]);

   int *sm = NULL;
   int *sm_d = NULL;

   sm = (int *) malloc(N * sizeof(int));

   hipMalloc((void**)&sm_d, N*sizeof(int));

   kern<<<N, N>>>(sm_d);

   hipMemcpy(sm, sm_d, N*sizeof(int), hipMemcpyDeviceToHost);

   for (int i=0;i<N;i++)

      printf("%d %d\n",i,sm[i]);

return 0;

}
